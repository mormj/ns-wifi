#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>


__global__ void multiply_const_kernel(hipFloatComplex *in, hipFloatComplex *out,
                                      hipFloatComplex k, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    // e ix = cos x + i sin x
    out[i] = hipCmulf(in[i], k);
  }
}



void exec_multiply_const(hipFloatComplex *in, hipFloatComplex *out,
                         hipFloatComplex k, int n, int grid_size, int block_size,
                         hipStream_t stream) {
  multiply_const_kernel<<<grid_size, block_size, 0, stream>>>(in, out, k, n);
}


void get_block_and_grid_multiply_const(int *minGrid, int *minBlock) {
  hipOccupancyMaxPotentialBlockSize(minGrid, minBlock, multiply_const_kernel,
                                     0, 0);
}