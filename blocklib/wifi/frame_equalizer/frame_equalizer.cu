#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>


__host__ __device__ double carg(const hipFloatComplex &z) {
  return atan2(hipCimagf(z), hipCrealf(z));
}
__host__ __device__ hipFloatComplex conj(const hipFloatComplex &z) {
  return make_hipFloatComplex(z.x, -z.y);
}

__global__ void calc_beta_err_kernel(hipFloatComplex *in, float *polarity,
                                     int current_symbol_index,
                                     hipFloatComplex *last_symbol, float bw,
                                     float freq, float *beta, float *err,
                                     int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < n) {
    hipFloatComplex *current_symbol = &in[i * 64];
    hipFloatComplex pp0, pp1, pp2, pp3;

    if (i > 0) {
      pp0 = in[(i - 1) * 64 + 11];
      pp1 = in[(i - 1) * 64 + 25];
      pp2 = in[(i - 1) * 64 + 39];
      pp3 = in[(i - 1) * 64 + 53];
    } else {
      pp0 = last_symbol[11];
      pp1 = last_symbol[25];
      pp2 = last_symbol[39];
      pp3 = last_symbol[53];
    }

    float p = polarity[(current_symbol_index + i - 2) % 127];

    if (current_symbol_index + i <= 2) {
      pp1 = hipCmulf(pp1, make_hipFloatComplex(-1.0, 0.0));
    } else {
      float last_p = polarity[(current_symbol_index + i - 2 - 1) % 127];
      pp0 = hipCmulf(pp0, make_hipFloatComplex(last_p, 0.0));
      pp1 = hipCmulf(pp1, make_hipFloatComplex(last_p, 0.0));
      pp2 = hipCmulf(pp2, make_hipFloatComplex(last_p, 0.0));
      pp3 = hipCmulf(pp3, make_hipFloatComplex(-last_p, 0.0));
    }

    if ((current_symbol_index + i) < 2) {
      beta[i] = carg(
          make_hipFloatComplex(current_symbol[11].x - current_symbol[25].x +
                                  current_symbol[39].x + current_symbol[53].x,
                              current_symbol[11].y - current_symbol[25].y +
                                  current_symbol[39].y + current_symbol[53].y));
    } else {
      beta[i] = carg(make_hipFloatComplex(
          (current_symbol[11].x * p) + (current_symbol[39].x * p) +
              (current_symbol[25].x * p) + (current_symbol[53].x * -p),
          (current_symbol[11].y * p) + (current_symbol[39].y * p) +
              (current_symbol[25].y * p) + (current_symbol[53].y * -p)));
    }

    err[i] = carg(hipCaddf(
        hipCaddf((make_hipFloatComplex(hipCmulf(pp0, current_symbol[11]).x * p,
                                     -hipCmulf(pp0, current_symbol[11]).y * p)),
                (make_hipFloatComplex(hipCmulf(pp1, current_symbol[25]).x * p,
                                     -hipCmulf(pp1, current_symbol[25]).y * p))),
        hipCaddf(
            (make_hipFloatComplex(hipCmulf(pp2, current_symbol[39]).x * p,
                                 -hipCmulf(pp2, current_symbol[39]).y * p)),
            (make_hipFloatComplex(hipCmulf(pp3, current_symbol[53]).x * -p,
                                 -hipCmulf(pp3, current_symbol[53]).y * -p)))));

    err[i] *= (bw / (2 * M_PI * freq * 80));
  }
}

__global__ void correct_sampling_offset_kernel(hipFloatComplex *in,
                                               hipFloatComplex *out,
                                               int start_idx, float freq_offset,
                                               int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < n) {

    int symbol_index = i / 64;
    int sample_index = i % 64;

    // // compensate sampling offset
    // for(int i = 0; i < 64; i++) {
    // 	current_symbol[i] *= exp(gr_complex(0,
    // 2*M_PI*d_current_symbol*80*(d_epsilon0 + d_er)*(i-32)/64));
    // }

    float x = -freq_offset * (float)(start_idx + symbol_index) *
              (float)(sample_index - 32) / 64;
    out[i] = hipCmulf(in[i], make_hipFloatComplex(cos(x), sin(x)));
  }
}

__global__ void multiply_phase_kernel(hipFloatComplex *in, hipFloatComplex *out,
                                      float *beta, int n) {
  
  // beta applies for an entire symbol
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    // e ix = cos x + i sin x
    out[i] = hipCmulf(in[i], make_hipFloatComplex(cos(beta[i/64]), sin(beta[i/64])));
  }
}

void exec_calc_beta_err(hipFloatComplex *in, float *polarity,
                        int current_symbol_index, hipFloatComplex *last_symbol,
                        float bw, float freq, float *beta, float *err, int n,
                        int grid_size, int block_size, hipStream_t stream) {
  calc_beta_err_kernel<<<grid_size, block_size, 0, stream>>>(
      in, polarity, current_symbol_index, last_symbol, bw, freq, beta, err, n);
}

void get_block_and_grid_calc_beta_err(int *minGrid, int *minBlock) {
  hipOccupancyMaxPotentialBlockSize(minGrid, minBlock, calc_beta_err_kernel, 0,
                                     0);
}

void exec_correct_sampling_offset(hipFloatComplex *in, hipFloatComplex *out,
                                  int start_idx, float freq_offset, int n,
                                  int grid_size, int block_size,
                                  hipStream_t stream) {
  correct_sampling_offset_kernel<<<grid_size, block_size, 0, stream>>>(
      in, out, start_idx, freq_offset, n);
}

void exec_multiply_phase(hipFloatComplex *in, hipFloatComplex *out, float *beta,
                         int n, int grid_size, int block_size,
                         hipStream_t stream) {
  multiply_phase_kernel<<<grid_size, block_size, 0, stream>>>(in, out, beta, n);
}