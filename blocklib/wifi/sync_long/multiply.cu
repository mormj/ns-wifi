#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>


__global__ void multiply_kernel_ccc(hipFloatComplex *in1, hipFloatComplex *in2,
                                    hipFloatComplex *out, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    float re, im;
    re = in1[i].x * in2[i].x - in1[i].y * in2[i].y;
    im = in1[i].x * in2[i].y + in1[i].y * in2[i].x;
    out[i].x = re;
    out[i].y = im;
  }
}

void exec_multiply_kernel_ccc(hipFloatComplex *in1, hipFloatComplex *in2,
                              hipFloatComplex *out, int n, int grid_size,
                              int block_size, hipStream_t stream) {
  multiply_kernel_ccc<<<grid_size, block_size, 0, stream>>>(in1, in2, out, n);
}

void get_block_and_grid_multiply(int *minGrid, int *minBlock) {
  hipOccupancyMaxPotentialBlockSize(minGrid, minBlock, multiply_kernel_ccc, 0,
                                     0);
}