#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>


__global__ void
remove_cp(hipFloatComplex* in, hipFloatComplex* out, int symlen, int cplen, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        int sym_idx = i / symlen;
        int samp_idx = i % symlen;

        if (samp_idx >= cplen) {
            out[sym_idx * (symlen-cplen) + samp_idx - cplen] = in[sym_idx * symlen + samp_idx];
        }
    }
}

void exec_remove_cp(hipFloatComplex* in,
                    hipFloatComplex* out,
                    int symlen,
                    int cplen,
                    int n,
                    int grid_size,
                    int block_size,
                    hipStream_t stream)
{
    remove_cp<<<grid_size, block_size, 0, stream>>>(in, out, symlen, cplen, n);
}

void get_block_and_grid_remove_cp(int* minGrid, int* minBlock)
{
    hipOccupancyMaxPotentialBlockSize(minGrid, minBlock, remove_cp, 0, 0);
}

__global__ void
remove_cp_freqcorr(hipFloatComplex* in, hipFloatComplex* out, int symlen, int cplen, int n, float freqoff, int start_sym)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        int sym_idx = i / symlen;
        int samp_idx = i % symlen;

        if (samp_idx >= cplen) {
            out[sym_idx * (symlen-cplen) + samp_idx - cplen] = in[sym_idx * symlen + samp_idx];
            float x = ((start_sym + sym_idx) * symlen + samp_idx) * freqoff;
            hipCmulf(in[i], make_hipFloatComplex(cos(x), sin(x)));
        }
    }
}

void exec_remove_cp_freqcorr(hipFloatComplex* in,
    hipFloatComplex* out,
    int symlen,
    int cplen,
    int n,
    int grid_size,
    int block_size,
    float freqoff,
    int start_sym,
    hipStream_t stream)
{
    remove_cp_freqcorr<<<grid_size, block_size, 0, stream>>>(in, out, symlen, cplen, n, freqoff, start_sym);
}